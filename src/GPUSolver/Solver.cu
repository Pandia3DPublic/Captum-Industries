#include "hip/hip_runtime.h"
#include <iostream>
#include <Cuda/Common/UtilsCuda.h>// for open3d cuda error check

////for debug purposes
//#define PRINT_RESIDUALS_SPARSE
//#define CUDA_ERROR_CHECK
//#define PRINT_RESIDUALS_DENSE

#define ENABLE_EARLY_OUT

#include "../GlobalDefines.h"
#include "SolverParameters.h"
#include "SolverState.h"
#include "SolverUtil.h"
#include "SolverEquations.h"
#include "CUDATimer.h"
//#include "SolverBundlingEquationsLie.h"
//#include "SolverBundlingDenseUtil.h"

#include <conio.h>

#define THREADS_PER_BLOCK_DENSE_DEPTH 128
#define THREADS_PER_BLOCK_DENSE_DEPTH_FLIP 64

#define THREADS_PER_BLOCK_DENSE_OVERLAP 512



__global__ void FlipJtJ_Kernel(unsigned int total, unsigned int dim, float* d_JtJ)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx < total) {
		const unsigned int x = idx % dim;
		const unsigned int y = idx / dim;
		if (x > y) {
			d_JtJ[y * dim + x] = d_JtJ[x * dim + y];
		}
	}
}

//####################### all commented out residual stuf that for some reason uses lie algebra thingys ###############

////todo more efficient?? (there are multiple per image-image...)
////get high residuals
//todo why does this need lie algebra stuff?
//__global__ void collectHighResidualsDevice(SolverInput input, SolverState state, SolverStateAnalysis analysis, SolverParameters parameters, unsigned int maxNumHighResiduals)
//{
//	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
//	const unsigned int corrIdx = blockIdx.x * blockDim.x + threadIdx.x;
//
//	if (corrIdx < N) {
//		float residual = evalAbsMaxResidualDevice(corrIdx, input, state, parameters);
//		if (residual > parameters.highResidualThresh) {
//			int idx = atomicAdd(state.d_countHighResidual, 1);
//			if (idx < maxNumHighResiduals) {
//				analysis.d_maxResidual[idx] = residual;
//				analysis.d_maxResidualIndex[idx] = corrIdx;
//			}
//		}
//	}
//}
//void collectHighResiduals(SolverInput& input, SolverState& state, SolverStateAnalysis& analysis, SolverParameters& parameters, CUDATimer* timer)
//{
//	if (timer) timer->startEvent(__FUNCTION__);
//	cutilSafeCall(hipMemset(state.d_countHighResidual, 0, sizeof(int)));
//
//	const unsigned int N = input.numberOfCorrespondences; // Number of correspondences 
//	unsigned int maxNumHighResiduals = (input.maxCorrPerImage*input.maxNumberOfImages + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
//	collectHighResidualsDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, analysis, parameters, maxNumHighResiduals);
//
//#ifdef _DEBUG
//	cutilSafeCall(hipDeviceSynchronize());
//	cutilCheckMsg(__FUNCTION__);
//#endif
//	if (timer) timer->endEvent();
//}

/////////////////////////////////////////////////////////////////////////
// Eval Max Residual
/////////////////////////////////////////////////////////////////////////

__global__ void EvalMaxResidualDevice(SolverInput input, SolverState state, SolverStateAnalysis analysis, SolverParameters parameters)
{
	__shared__ int maxResIndex[THREADS_PER_BLOCK];
	__shared__ float maxRes[THREADS_PER_BLOCK];

	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	const unsigned int corrIdx = blockIdx.x * blockDim.x + threadIdx.x;

	maxResIndex[threadIdx.x] = 0;
	maxRes[threadIdx.x] = 0.0f;

	if (corrIdx < N) {
		float residual = evalAbsMaxResidualDevice(corrIdx, input, state, parameters);

		maxRes[threadIdx.x] = residual;
		maxResIndex[threadIdx.x] = corrIdx;

		__syncthreads();

		for (int stride = THREADS_PER_BLOCK / 2; stride > 0; stride /= 2) {

			if (threadIdx.x < stride) {
				int first = threadIdx.x;
				int second = threadIdx.x + stride;
				if (maxRes[first] < maxRes[second]) {
					maxRes[first] = maxRes[second];
					maxResIndex[first] = maxResIndex[second];
				}
			}

			__syncthreads();
		}

		if (threadIdx.x == 0) {
			//printf("d_maxResidual[%d] = %f (index %d)\n", blockIdx.x, maxRes[0], maxResIndex[0]);
			analysis.d_maxResidual[blockIdx.x] = maxRes[0];
			analysis.d_maxResidualIndex[blockIdx.x] = maxResIndex[0];
		}
	}
}

void evalMaxResidual(SolverInput& input, SolverState& state, SolverStateAnalysis& analysis, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	const unsigned int N = input.numberOfCorrespondences; // Number of correspondences 
	EvalMaxResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, analysis, parameters);

#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif

	if (timer) timer->endEvent();
}

/////////////////////////////////////////////////////////////////////////
// Eval Cost
/////////////////////////////////////////////////////////////////////////

__global__ void ResetResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	if (x == 0) state.d_sumResidual[0] = 0.0f;
}

__global__ void EvalResidualDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences; // Number of cors
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float residual = 0.0f;
	if (x < N) {
		residual = evalFDevice(x, input, state, parameters,false);
		//float out = warpReduce(residual);
		//unsigned int laneid;
		////This command gets the lane ID within the current warp
		//asm("mov.u32 %0, %%laneid;" : "=r"(laneid));
		//if (laneid == 0) {
		//	atomicAdd(&state.d_sumResidual[0], out);
		//}
		atomicAdd(&state.d_sumResidual[0], residual);
	}
}

float EvalResidual(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	float residual = 0.0f;

	const unsigned int N = input.numberOfCorrespondences; // Number of cors
	ResetResidualDevice << < 1, 1, 1 >> >(input, state, parameters);
	EvalResidualDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);

	residual = state.getSumResidual();

#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif


	if (timer) timer->endEvent();

	return residual;
}

/////////////////////////////////////////////////////////////////////////
// Count High Residuals
/////////////////////////////////////////////////////////////////////////
//
__global__ void CountHighResidualsDevice(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences; // Number of block variables
	const unsigned int corrIdx = blockIdx.x * blockDim.x + threadIdx.x;

	if (corrIdx < N) {
		float residual = evalAbsMaxResidualDevice(corrIdx, input, state, parameters);

		if (residual > parameters.verifyOptDistThresh)
			atomicAdd(state.d_countHighResidual, 1);
	}
}

int countHighResiduals(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	const unsigned int N = input.numberOfCorrespondences; // Number of correspondences
	cutilSafeCall(hipMemset(state.d_countHighResidual, 0, sizeof(int)));
	CountHighResidualsDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(input, state, parameters);

	int count;
	CheckCuda(hipMemcpy(&count, state.d_countHighResidual, sizeof(int), hipMemcpyDeviceToHost));
#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif


	if (timer) timer->endEvent();
	return count;
}

/////////////////////////////////////////////////////////////////////////
// Convergence Analysis
/////////////////////////////////////////////////////////////////////////

//uses same data store as max residual
__global__ void EvalGNConvergenceDevice(SolverInput input, SolverStateAnalysis analysis, SolverState state) //compute max of delta
{
	__shared__ float maxVal[THREADS_PER_BLOCK];

	const unsigned int N = input.numberOfImages;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	maxVal[threadIdx.x] = 0.0f;

	if (x < N)
	{
		if (x == 0)
			maxVal[threadIdx.x] = 0.0f;
		else {
			float3 r3 = fmaxf(fabs(state.d_deltaRot[x]), fabs(state.d_deltaTrans[x]));
			float r = fmaxf(r3.x, fmaxf(r3.y, r3.z));
			maxVal[threadIdx.x] = r;
		}
		__syncthreads();

		for (int stride = THREADS_PER_BLOCK / 2; stride > 0; stride /= 2) {
			if (threadIdx.x < stride) {
				int first = threadIdx.x;
				int second = threadIdx.x + stride;
				maxVal[first] = fmaxf(maxVal[first], maxVal[second]);
			}
			__syncthreads();
		}
		if (threadIdx.x == 0) {
			analysis.d_maxResidual[blockIdx.x] = maxVal[0];
		}
	}
}

float EvalGNConvergence(SolverInput& input, SolverState& state, SolverStateAnalysis& analysis, CUDATimer* timer)
{
	if (timer) timer->startEvent(__FUNCTION__);

	const unsigned int N = input.numberOfImages;
	const unsigned int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
	EvalGNConvergenceDevice << < blocksPerGrid, THREADS_PER_BLOCK >> >(input, analysis, state);

#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif

	//copy to host and compute max
	CheckCuda(hipMemcpy(analysis.h_maxResidual, analysis.d_maxResidual, sizeof(float) * blocksPerGrid, hipMemcpyDeviceToHost));
	CheckCuda(hipMemcpy(analysis.h_maxResidualIndex, analysis.d_maxResidualIndex, sizeof(int) * blocksPerGrid, hipMemcpyDeviceToHost));
	float maxVal = 0.0f;
	for (unsigned int i = 0; i < blocksPerGrid; i++) {
		if (maxVal < analysis.h_maxResidual[i]) maxVal = analysis.h_maxResidual[i];
	}
	if (timer) timer->endEvent();

	return maxVal;
}

// For the naming scheme of the variables see:
// http://en.wikipedia.org/wiki/Conjugate_gradient_method
// This code is an implementation of their PCG pseudo code

template<bool useDense>
__global__ void PCGInit_Kernel1(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;
	const int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x > 0 && x < N)
	{
		float3 resRot, resTrans;
		evalMinusJTFDevice<useDense>(x, input, state, parameters, resRot, resTrans);  // residuum = J^T x -F - A x delta_0  => J^T x -F, since A x x_0 == 0 

		state.d_rRot[x] = resRot;											// store for next iteration
		state.d_rTrans[x] = resTrans;										// store for next iteration

		const float3 pRot = state.d_precondionerRot[x] * resRot;			// apply preconditioner M^-1
		state.d_pRot[x] = pRot;

		const float3 pTrans = state.d_precondionerTrans[x] * resTrans;		// apply preconditioner M^-1
		state.d_pTrans[x] = pTrans;

		d = dot(resRot, pRot) + dot(resTrans, pTrans);						// x-th term of nomimator for computing alpha and denominator for computing beta

		state.d_Ap_XRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_Ap_XTrans[x] = make_float3(0.0f, 0.0f, 0.0f);
	}

	d = warpReduce(d);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(state.d_scanAlpha, d);
	}
}

__global__ void PCGInit_Kernel2(unsigned int N, SolverState state)
{
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x > 0 && x < N) state.d_rDotzOld[x] = state.d_scanAlpha[0];				// store result for next kernel call
}

void Initialization(SolverInput& input, SolverState& state, SolverParameters& parameters, CUDATimer* timer)
{
	const unsigned int N = input.numberOfImages;

	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}

	if (timer) timer->startEvent("Initialization");

	//!!!DEBUGGING //remember to uncomment the delete...
	//float3* rRot = new float3[input.numberOfImages]; // -jtf
	//float3* rTrans = new float3[input.numberOfImages];
	//!!!DEBUGGING

	CheckCuda(hipMemset(state.d_scanAlpha, 0, sizeof(float)));
#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif

	
	PCGInit_Kernel1<false> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif


	PCGInit_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK >> >(N, state);
#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif

	if (timer) timer->endEvent();

	//float scanAlpha;
	//cutilSafeCall(hipMemcpy(&scanAlpha, state.d_scanAlpha, sizeof(float), hipMemcpyDeviceToHost));
	//if (rRot) delete[] rRot;
	//if (rTrans) delete[] rTrans;
}

/////////////////////////////////////////////////////////////////////////
// PCG Iteration Parts
/////////////////////////////////////////////////////////////////////////

__global__ void PCGStep_Kernel0(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfCorrespondences;					// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N)
	{
		const float3 tmp = applyJDevice(x, input, state, parameters);		// A x p_k  => J^T x J x p_k 
		state.d_Jp[x] = tmp;												// store for next kernel call
	}
}

__global__ void PCGStep_Kernel1a(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;							// Number of block variables
	const unsigned int x = blockIdx.x;
	const unsigned int lane = threadIdx.x % WARP_SIZE;

	if (x > 0 && x < N)
	{
		float3 rot, trans;
		applyJTDevice(x, input, state, parameters, rot, trans, threadIdx.x, lane);			// A x p_k  => J^T x J x p_k 

		if (lane == 0)
		{
			atomicAdd(&state.d_Ap_XRot[x].x, rot.x);
			atomicAdd(&state.d_Ap_XRot[x].y, rot.y);
			atomicAdd(&state.d_Ap_XRot[x].z, rot.z);

			atomicAdd(&state.d_Ap_XTrans[x].x, trans.x);
			atomicAdd(&state.d_Ap_XTrans[x].y, trans.y);
			atomicAdd(&state.d_Ap_XTrans[x].z, trans.z);
		}
	}
}

__global__ void PCGStep_Kernel1b(SolverInput input, SolverState state, SolverParameters parameters)
{
	const unsigned int N = input.numberOfImages;								// Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	float d = 0.0f;
	if (x > 0 && x < N)
	{
		d = dot(state.d_pRot[x], state.d_Ap_XRot[x]) + dot(state.d_pTrans[x], state.d_Ap_XTrans[x]);		// x-th term of denominator of alpha
	}

	d = warpReduce(d);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(state.d_scanAlpha, d);
	}
}

__global__ void PCGStep_Kernel2(SolverInput input, SolverState state)
{
	const unsigned int N = input.numberOfImages;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	const float dotProduct = state.d_scanAlpha[0];

	float b = 0.0f;
	if (x > 0 && x < N)
	{
		float alpha = 0.0f;
		if (dotProduct > FLOAT_EPSILON) alpha = state.d_rDotzOld[x] / dotProduct;		// update step size alpha

		state.d_deltaRot[x] = state.d_deltaRot[x] + alpha*state.d_pRot[x];			// do a decent step
		state.d_deltaTrans[x] = state.d_deltaTrans[x] + alpha*state.d_pTrans[x];	// do a decent step

		float3 rRot = state.d_rRot[x] - alpha*state.d_Ap_XRot[x];					// update residuum
		state.d_rRot[x] = rRot;														// store for next kernel call

		float3 rTrans = state.d_rTrans[x] - alpha*state.d_Ap_XTrans[x];				// update residuum
		state.d_rTrans[x] = rTrans;													// store for next kernel call

		float3 zRot = state.d_precondionerRot[x] * rRot;							// apply preconditioner M^-1
		state.d_zRot[x] = zRot;														// save for next kernel call

		float3 zTrans = state.d_precondionerTrans[x] * rTrans;						// apply preconditioner M^-1
		state.d_zTrans[x] = zTrans;													// save for next kernel call

		b = dot(zRot, rRot) + dot(zTrans, rTrans);									// compute x-th term of the nominator of beta
	}
	b = warpReduce(b);
	if (threadIdx.x % WARP_SIZE == 0)
	{
		atomicAdd(&state.d_scanAlpha[1], b);
	}
}

template<bool lastIteration>
__global__ void PCGStep_Kernel3(SolverInput input, SolverState state)
{
	const unsigned int N = input.numberOfImages;
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x > 0 && x < N)
	{
		const float rDotzNew = state.d_scanAlpha[1];								// get new nominator
		const float rDotzOld = state.d_rDotzOld[x];								// get old denominator

		float beta = 0.0f;
		if (rDotzOld > FLOAT_EPSILON) beta = rDotzNew / rDotzOld;				// update step size beta

		state.d_rDotzOld[x] = rDotzNew;											// save new rDotz for next iteration
		state.d_pRot[x] = state.d_zRot[x] + beta*state.d_pRot[x];		// update decent direction
		state.d_pTrans[x] = state.d_zTrans[x] + beta*state.d_pTrans[x];		// update decent direction


		state.d_Ap_XRot[x] = make_float3(0.0f, 0.0f, 0.0f);
		state.d_Ap_XTrans[x] = make_float3(0.0f, 0.0f, 0.0f);

		if (lastIteration)
		{
			//if (input.d_validImages[x]) { //not really necessary
#ifdef USE_LIE_SPACE //TODO just keep that matrix transforms around
			float3 rot, trans;
			computeLieUpdate(state.d_deltaRot[x], state.d_deltaTrans[x], state.d_xRot[x], state.d_xTrans[x], rot, trans);
			state.d_xRot[x] = rot;
			state.d_xTrans[x] = trans;
#else
			state.d_xRot[x] = state.d_xRot[x] + state.d_deltaRot[x];
			state.d_xTrans[x] = state.d_xTrans[x] + state.d_deltaTrans[x];
#endif
			//}
		}
	}
}

template<bool useSparse, bool useDense>
bool PCGIteration(SolverInput& input, SolverState& state, SolverParameters& parameters, SolverStateAnalysis& analysis, bool lastIteration, CUDATimer *timer)
{
	const unsigned int N = input.numberOfImages;	// Number of block variables

	// Do PCG step
	const int blocksPerGrid = (N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

	if (blocksPerGrid > THREADS_PER_BLOCK)
	{
		std::cout << "Too many variables for this block size. Maximum number of variables for two kernel scan: " << THREADS_PER_BLOCK*THREADS_PER_BLOCK << std::endl;
		while (1);
	}
	if (timer) timer->startEvent("PCGIteration");

	CheckCuda(hipMemset(state.d_scanAlpha, 0, sizeof(float) * 2));

	// sparse part
	if (useSparse) {
		const unsigned int Ncorr = input.numberOfCorrespondences;
		const int blocksPerGridCorr = (Ncorr + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
		PCGStep_Kernel0 << <blocksPerGridCorr, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef CUDA_ERROR_CHECK
		CheckCuda(hipDeviceSynchronize());
		CheckCuda(hipGetLastError());
#endif

		PCGStep_Kernel1a << < N, THREADS_PER_BLOCK_JT >> >(input, state, parameters);
#ifdef CUDA_ERROR_CHECK
		CheckCuda(hipDeviceSynchronize());
		CheckCuda(hipGetLastError());
#endif

	}

	PCGStep_Kernel1b << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state, parameters);
#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif

	PCGStep_Kernel2 << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif

#ifdef ENABLE_EARLY_OUT //for convergence
	float scanAlpha; CheckCuda(hipMemcpy(&scanAlpha, state.d_scanAlpha, sizeof(float), hipMemcpyDeviceToHost));
	//if (fabs(scanAlpha) < 0.00005f) lastIteration = true;  //todo check this part
	//if (fabs(scanAlpha) < 1e-6) lastIteration = true;  //todo check this part
	if (fabs(scanAlpha) < 5e-7) { lastIteration = true; }  //todo check this part
#endif
	if (lastIteration) {
		PCGStep_Kernel3<true> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
	}
	else {
		PCGStep_Kernel3<false> << <blocksPerGrid, THREADS_PER_BLOCK >> >(input, state);
	}

#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif
	if (timer) timer->endEvent();

	return lastIteration;
}

////////////////////////////////////////////////////////////////////
// Main GN Solver Loop
////////////////////////////////////////////////////////////////////

void solveBundlingStub(SolverInput& input, SolverState& state, SolverParameters& parameters, SolverStateAnalysis& analysis, float* convergenceAnalysis, CUDATimer *timer)
{
	if (convergenceAnalysis) { //this only occurs if m_bRecordConvergence is set to true in constructor
		float initialResidual = EvalResidual(input, state, parameters, timer);
		convergenceAnalysis[0] = initialResidual; // initial residual
	}

	//!!!DEBUGGING
#ifdef PRINT_RESIDUALS_SPARSE
	if (parameters.weightSparse > 0) {
		if (input.numberOfCorrespondences == 0) { printf("ERROR: %d correspondences\n", input.numberOfCorrespondences); getchar(); }
		float initialResidual = EvalResidual(input, state, parameters, timer);
		printf("initial sparse = %f*%f = %f\n", parameters.weightSparse, initialResidual / parameters.weightSparse, initialResidual);
		//note: for some stupid reason  parameters.weightSparse is just the first element of the sparse weight vector 
	}
#endif


	for (unsigned int nIter = 0; nIter < parameters.nNonLinearIterations; nIter++)
	{
		parameters.weightSparse = input.weightsSparse[nIter];

		//probably builds the local linear problem
		Initialization(input, state, parameters, timer);

		if (parameters.weightSparse > 0.0f) {
			for (unsigned int linIter = 0; linIter < parameters.nLinIterations; linIter++){
				if (PCGIteration<true, false>(input, state, parameters, analysis, linIter == parameters.nLinIterations - 1, timer)) {
					//totalLinIters += (linIter+1); numLin++; 
					break;
				}
			}
		}
		else {
			std::cout << "weight is zero, something is weird in solver.cu line 620 \n";
		}

#ifdef PRINT_RESIDUALS_SPARSE
		if (parameters.weightSparse > 0) {
			float residual = EvalResidual(input, state, parameters, timer);
			printf("[niter %d] weight * sparse = %f*%f = %f\t[#corr = %d]\n", nIter, parameters.weightSparse, residual / parameters.weightSparse, residual, input.numberOfCorrespondences);
		}
#endif
		if (convergenceAnalysis) {
			float residual = EvalResidual(input, state, parameters, timer);
			convergenceAnalysis[nIter + 1] = residual;
		}

		//if (timer) timer->evaluate(true);

#ifdef ENABLE_EARLY_OUT //convergence
		//if (nIter < parameters.nNonLinearIterations - 1 && EvalGNConvergence(input, state, analysis, timer) < 0.01f) { //!!! TODO CHECK HOW THESE GENERALIZE
		if (nIter < parameters.nNonLinearIterations - 1 && EvalGNConvergence(input, state, analysis, timer) < 0.005f) { //0.001?
		//if (nIter < parameters.nNonLinearIterations - 1 && EvalGNConvergence(input, state, analysis, timer) < 0.001f) { 
			//if (!parameters.useDense) { totalNonLinIters += (nIter+1); numNonLin++; }
#ifdef PRINT_RESIDUALS_SPARSE
			std::cout << "stopped due to convergence analysis \n";
#endif
			break;
		}
		//else if (!parameters.useDense && nIter == parameters.nNonLinearIterations - 1) { totalNonLinIters += (nIter+1); numNonLin++; }
#endif
		}

	}

////////////////////////////////////////////////////////////////////
// build variables to correspondences lookup
////////////////////////////////////////////////////////////////////

__global__ void BuildVariablesToCorrespondencesTableDevice(EntryJ* d_correspondences, unsigned int numberOfCorrespondences,
	unsigned int maxNumCorrespondencesPerImage, int* d_variablesToCorrespondences, int* d_numEntriesPerRow)
{
	const unsigned int N = numberOfCorrespondences; // Number of block variables
	const unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;

	if (x < N) {
		EntryJ& corr = d_correspondences[x];
		if (corr.isValid()) {
			int offset0 = atomicAdd(&d_numEntriesPerRow[corr.imgIdx_i], 1); // may overflow - need to check when read
			int offset1 = atomicAdd(&d_numEntriesPerRow[corr.imgIdx_j], 1); // may overflow - need to check when read
			if (offset0 < maxNumCorrespondencesPerImage && offset1 < maxNumCorrespondencesPerImage)	{
				d_variablesToCorrespondences[corr.imgIdx_i * maxNumCorrespondencesPerImage + offset0] = x;
				d_variablesToCorrespondences[corr.imgIdx_j * maxNumCorrespondencesPerImage + offset1] = x;
			}
			else { //invalidate
				printf("EXCEEDED MAX NUM CORR PER IMAGE IN SOLVER, INVALIDATING %d(%d,%d) [%d,%d | %d]\n",
					x, corr.imgIdx_i, corr.imgIdx_j, offset0, offset1, maxNumCorrespondencesPerImage); //debugging
				corr.setInvalid(); //make sure j corresponds to jt
			}
		}
	}
}

void buildVariablesToCorrespondencesTableCUDA(EntryJ* d_correspondences, unsigned int numberOfCorrespondences, unsigned int maxNumCorrespondencesPerImage, int* d_variablesToCorrespondences, int* d_numEntriesPerRow, CUDATimer* timer)
{
	const unsigned int N = numberOfCorrespondences;

	if (timer) timer->startEvent(__FUNCTION__);

	BuildVariablesToCorrespondencesTableDevice << <(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >> >(d_correspondences, numberOfCorrespondences, maxNumCorrespondencesPerImage, d_variablesToCorrespondences, d_numEntriesPerRow);
#ifdef CUDA_ERROR_CHECK
	CheckCuda(hipDeviceSynchronize());
	CheckCuda(hipGetLastError());
#endif

	if (timer) timer->endEvent();
}
